#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int arraysize = 121;
	float train[121][2];
	float trainOutput[121];
	//float Centers[][] = { { -1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1,-1,-0.800000000000000,-0.600000000000000,-0.400000000000000,-0.200000000000000,0,0.200000000000000,0.400000000000000,0.600000000000000,0.800000000000000,1 }
	//{-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.800000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.600000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.400000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,-0.200000000000000,0,0,0,0,0,0,0,0,0,0,0,0.200000000000000,0.200000000000000,0.200000000000000,0.200000000000000,0.200000000000000,0.200000000000000,0.200000000000000,0.200000000000000,0.200000000000000,0.200000000000000,0.200000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.400000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.600000000000000,0.800000000000000,0.800000000000000,0.800000000000000,0.800000000000000,0.800000000000000,0.800000000000000,0.800000000000000,0.800000000000000,0.800000000000000,0.800000000000000,0.800000000000000,1,1,1,1,1,1,1,1,1,1,1} };
	//float train[11];
	float data[11];
	data[0] = -1.0;
	for (int i = 1; i < 11; i++)
	{
		data[i] = data[i - 1] + 0.2;
		printf("%f\n", data[i]);
	}
	int count = 0;
	for (int i = 0; i < 11; i++)
	{
		for (int j = 0; j < 11; j++)
		{
			train[count][0] = data[j];
			train[count][1] = data[i];
			trainOutput[count] = exp(-pow(train[count][0], 2) - train[count][1]);
			printf("%.2f, %.2f %.2f\n", train[count][0], train[count][1], trainOutput[count]);
			count++;
		}
	}
	






	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
